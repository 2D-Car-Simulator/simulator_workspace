#include "hip/hip_runtime.h"
#include "parallel_cost_calculations.cuh"

namespace cuda_mp
{

__global__
void calculate_cost()
{
    printf("\n Thread id printing from GPU: %d ", threadIdx.x);
}

void calculateCost(const mp::Node* node)
{
    calculate_cost <<<2,2>>> ();
}

}
