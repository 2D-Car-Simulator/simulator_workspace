#include "hip/hip_runtime.h"
#include "parallel_cost_calculations.cuh"

namespace cuda_mp
{

__global__
void calculate_cost(mp::Node* device_node_array)
{
    printf("\n Theta: %f, Pose: %f , %f ", device_node_array[threadIdx.x].pose.theta, device_node_array[threadIdx.x].pose.x, device_node_array[threadIdx.x].pose.y);
}

void calculateCost(std::vector<mp::Node>& child_nodes, const mp::PlannerConfig& config, const std::shared_ptr<mp::OverallInfo>& overall_info)
{
    //convert to vector to array
    mp::Node* host_node_array = child_nodes.data();

    uint nodeSize = sizeof (child_nodes[0]);
    uint totalNodes = child_nodes.size();
    uint totalNodesSize = totalNodes * nodeSize;

    mp::Node* device_node_array;

    hipMalloc ((void**)&device_node_array, totalNodesSize);
    if (hipGetLastError () != hipSuccess)
        std::cout<<"Node array hipMalloc booommm!!!!"<<std::endl;

    hipMemcpy (device_node_array, host_node_array, totalNodesSize, hipMemcpyHostToDevice);
    if (hipGetLastError () != hipSuccess)
        std::cout<<"Node array memCopy booommm!!!!"<<std::endl;

    calculate_cost <<<1,child_nodes.size()>>> (device_node_array);
    hipDeviceSynchronize();

    hipMemcpy (host_node_array, device_node_array, totalNodesSize, hipMemcpyDeviceToHost);

    // Free memory on the gpu
    hipFree (device_node_array);

    // Store nodes evaluated in the recent cycle
    overall_info->mp_info.curr_eval_nodes.clear();
    for (int i = 0; i < totalNodes; ++i)
    {
        overall_info->mp_info.curr_eval_nodes.push_back (host_node_array[i]);
    }
}

}
