#include "hip/hip_runtime.h"
#include "parallel_cost_calculations.cuh"

namespace cuda_mp
{

__device__
void calculateVehiclePolygon(const mp::Node& node, mp::Vec2D* veh_polygon_points)
{
    double yaw = node.pose.theta;

    double front      = 3.869;    // approx (vehicle.length/2 + vehicle.wheel_base/2)
    double rear       = 0.910;    // approx (vehicle.length/2 - vehicle.wheel_base/2)
    double width_by_2 = 1.05;

    mp::Vec2D front_left;
    front_left.x = cos(yaw)*front + cos(yaw + M_PI_2)*width_by_2 + node.pose.x;
    front_left.y = sin(yaw)*front + sin(yaw + M_PI_2)*width_by_2 + node.pose.y;

    mp::Vec2D front_right;
    front_right.x = cos(yaw)*front + cos(yaw - M_PI_2)*width_by_2 + node.pose.x;
    front_right.y = sin(yaw)*front + sin(yaw - M_PI_2)*width_by_2 + node.pose.y;

    mp::Vec2D rear_left;
    rear_left.x = - cos(yaw)*rear + cos(yaw + M_PI_2)*width_by_2 + node.pose.x;
    rear_left.y = - sin(yaw)*rear + sin(yaw + M_PI_2)*width_by_2 + node.pose.y;

    mp::Vec2D rear_right;
    rear_right.x = - cos(yaw)*rear + cos(yaw - M_PI_2)*width_by_2 + node.pose.x;
    rear_right.y = - sin(yaw)*rear + sin(yaw - M_PI_2)*width_by_2 + node.pose.y;

    veh_polygon_points[0] = rear_left;
    veh_polygon_points[1] = front_left;
    veh_polygon_points[2] = front_right;
    veh_polygon_points[3] = rear_right;
    veh_polygon_points[4] = rear_left;
}

__device__
bool isPointInsidePolygon(mp::Vec2D* polygon, const mp::Node& node, const mp::Vec2D& point)
{
    calculateVehiclePolygon(node, polygon);



    return true;
}

__global__
void calculate_cost(mp::Node* device_node_array)
{
    printf("\n Theta: %f, Pose: %f , %f ", device_node_array[threadIdx.x].pose.theta, device_node_array[threadIdx.x].pose.x, device_node_array[threadIdx.x].pose.y);

    mp::Vec2D polygon[5];

    mp::Vec2D test;
    test.x = 25.0;
    test.y = 25.0;

    isPointInsidePolygon (polygon, device_node_array[threadIdx.x], test);
}

void calculateCost(std::vector<mp::Node>& child_nodes, const mp::PlannerConfig& config, const std::shared_ptr<mp::OverallInfo>& overall_info)
{
    //convert to vector to array
    mp::Node* host_node_array = child_nodes.data();

    uint nodeSize = sizeof (child_nodes[0]);
    uint totalNodes = child_nodes.size();
    uint totalNodesSize = totalNodes * nodeSize;

    mp::Node* device_node_array;

    hipMalloc ((void**)&device_node_array, totalNodesSize);
    if (hipGetLastError () != hipSuccess)
        std::cout<<"Node array hipMalloc booommm!!!!"<<std::endl;

    hipMemcpy (device_node_array, host_node_array, totalNodesSize, hipMemcpyHostToDevice);
    if (hipGetLastError () != hipSuccess)
        std::cout<<"Node array memCopy booommm!!!!"<<std::endl;

    calculate_cost <<<1,child_nodes.size()>>> (device_node_array);
    hipDeviceSynchronize();

    hipMemcpy (host_node_array, device_node_array, totalNodesSize, hipMemcpyDeviceToHost);

    // Free memory on the gpu
    hipFree (device_node_array);

    // Store nodes evaluated in the recent cycle
    overall_info->mp_info.curr_eval_nodes.clear();
    for (uint i = 0; i < totalNodes; ++i)
    {
        overall_info->mp_info.curr_eval_nodes.push_back (host_node_array[i]);
    }
}

}
