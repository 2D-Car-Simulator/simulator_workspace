#include "hip/hip_runtime.h"
#include "parallel_cost_calculations.cuh"

namespace cuda_mp
{

__global__
void calculate_cost(mp::Node* device_node_array)
{
    printf("\n Steering angle: %f ", device_node_array[threadIdx.x].steering);
}

void calculateCost(mp::Node* node, const mp::PlannerConfig& config)
{
    std::vector<mp::Node> child_nodes = mp::getChildNodes(node, config);

    //convert to vector to array
    mp::Node* host_node_array = child_nodes.data();

    uint nodeSize = sizeof (mp::Node);
    uint totalNodesSize = child_nodes.size() * nodeSize;

    mp::Node* device_node_array;

    hipMalloc ((void**)&device_node_array, totalNodesSize);
    if (hipGetLastError () != hipSuccess)
        std::cout<<"Node array hipMalloc booommm!!!!"<<std::endl;

    hipMemcpy (device_node_array, host_node_array, totalNodesSize, hipMemcpyHostToDevice);
    if (hipGetLastError () != hipSuccess)
        std::cout<<"Node array memCopy booommm!!!!"<<std::endl;

    calculate_cost <<<1,child_nodes.size()>>> (device_node_array);
}

}
