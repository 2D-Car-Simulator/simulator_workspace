#include "hip/hip_runtime.h"
#include "parallel_cost_calculations.cuh"

namespace cuda_mp
{

/**********************************DEVICE FUNCTIONS***************************************/

__device__
double Angle2D(double x1, double y1, double x2, double y2)
{
   double theta_1, theta_2;
   double dtheta;

   theta_1 = atan2(y1,x1);
   theta_2 = atan2(y2,x2);

   dtheta = theta_2 - theta_1;

   while (dtheta > M_PI)
      dtheta -= 2*M_PI;

   while (dtheta < -M_PI)
      dtheta += 2*M_PI;

   return(dtheta);
}

__device__
bool isPointInsidePolygon(const mp::Vec2D& point, const mp::Vec2D* polygon, const uint polygon_point_count)
{
    double angle = 0.0;

    mp::Vec2D p1, p2;

    for (uint i = 0; i < polygon_point_count; ++i)
    {
        p1.x = polygon[i].x - point.x;
        p1.y = polygon[i].y - point.y;

        p2.x = polygon[(i+1)%polygon_point_count].x - point.x;
        p2.y = polygon[(i+1)%polygon_point_count].y - point.y;

        angle += Angle2D (p1.x, p1.y, p2.x, p2.y);
    }

    if (fabs(angle) < M_PI)
        return false;

    return true;
}

__device__
bool isNodeInsideRoad(const mp::Vec2D* child_node_polygon_points, const mp::Vec2D* road_polygon, const uint road_polygon_point_count)
{
    bool inside = true;

    for (uint i=0; i < 4; ++i)
    {
        inside = isPointInsidePolygon (child_node_polygon_points[i], road_polygon, road_polygon_point_count);

        // Exit loop if even one point is outside the road polygon
        if (!inside)
        {
            break;
        }
    }

    return inside;
}

__device__
bool nodeCollidesWithTraffic(const mp::Vec2D* child_node_polygon_points, const mp::Vec2D* traffic_polygons, const uint traffic_veh_count)
{
    bool is_colliding = false;

    for (uint veh_num = 0; veh_num < traffic_veh_count; ++veh_num)
    {
        for (uint i = 0; i < 4; ++i)
        {
            is_colliding = isPointInsidePolygon(child_node_polygon_points[i],
                                                (traffic_polygons + 4*veh_num), 4);

            if (is_colliding)
                return true;
        }
    }

    return false;
}

__device__
void calculateVehiclePolygon(const mp::Node& node, mp::Vec2D* veh_polygon_points)
{
    double yaw = node.pose.theta;

    double front      = 3.869;    // approx (vehicle.length/2 + vehicle.wheel_base/2)
    double rear       = 0.910;    // approx (vehicle.length/2 - vehicle.wheel_base/2)
    double width_by_2 = 1.05;

    mp::Vec2D front_left;
    front_left.x = cos(yaw)*front + cos(yaw + M_PI_2)*width_by_2 + node.pose.x;
    front_left.y = sin(yaw)*front + sin(yaw + M_PI_2)*width_by_2 + node.pose.y;

    mp::Vec2D front_right;
    front_right.x = cos(yaw)*front + cos(yaw - M_PI_2)*width_by_2 + node.pose.x;
    front_right.y = sin(yaw)*front + sin(yaw - M_PI_2)*width_by_2 + node.pose.y;

    mp::Vec2D rear_left;
    rear_left.x = - cos(yaw)*rear + cos(yaw + M_PI_2)*width_by_2 + node.pose.x;
    rear_left.y = - sin(yaw)*rear + sin(yaw + M_PI_2)*width_by_2 + node.pose.y;

    mp::Vec2D rear_right;
    rear_right.x = - cos(yaw)*rear + cos(yaw - M_PI_2)*width_by_2 + node.pose.x;
    rear_right.y = - sin(yaw)*rear + sin(yaw - M_PI_2)*width_by_2 + node.pose.y;

    veh_polygon_points[0] = rear_left;
    veh_polygon_points[1] = front_left;
    veh_polygon_points[2] = front_right;
    veh_polygon_points[3] = rear_right;
}

__global__
void calculate_cost(std::size_t threads_per_block,
                    mp::Node* device_child_node_array,
                    mp::Vec2D* road_polygon, const uint road_polygon_point_count,
                    mp::Vec2D* traffic_polygons, const uint traffic_veh_count,
                    const double lane_center_y,
                    const double dist_to_goal,
                    const double start_x,
                    const double start_y)
{
    std::size_t curr_index = threadIdx.x + threads_per_block * blockIdx.x;
    mp::Node* curr_child_node = &device_child_node_array[curr_index];

    mp::Vec2D child_node_polygon_points[4];
    calculateVehiclePolygon( *curr_child_node, child_node_polygon_points );

    // Calculate cost of going off road
    bool inside_road_boundary = isNodeInsideRoad(child_node_polygon_points, road_polygon, road_polygon_point_count);

    if (!inside_road_boundary)
        curr_child_node->hx += 50;

    // Calculate cost of collision
    bool is_colliding = nodeCollidesWithTraffic (child_node_polygon_points, traffic_polygons, traffic_veh_count);

    if (is_colliding)
        curr_child_node->hx += 150;

    // Calculate cost of lane offset
    curr_child_node->hx += fabs(curr_child_node->pose.y - lane_center_y) * 20;

    double dist_sq = pow((curr_child_node->pose.x - start_x), 2) +
            pow((curr_child_node->pose.y - start_y), 2);

    double dist_from_start = pow(dist_sq, 0.5);

    // Distance from goal heuristic
//    curr_child_node->hx += dist_to_goal - dist_from_start;

    if (!inside_road_boundary || is_colliding)
        curr_child_node->safe = false;
}

/***********************************HOST FUNCTIONS****************************************/

void checkCudaError(const std::string& error_msg)
{
    if (hipGetLastError () != hipSuccess)
        std::cout<<error_msg<<std::endl;
}

void calculateCost(std::vector<mp::Node>& child_nodes, const mp::PlannerConfig& config, const std::shared_ptr<mp::OverallInfo>& overall_info)
{
    //convert to child node vector to array
    mp::Node* host_child_node_array = child_nodes.data();

    uint totalChildNodes = child_nodes.size();
    uint totalChildNodesSize = totalChildNodes * sizeof(mp::Node);

    mp::Node* device_child_node_array;

    hipMalloc ((void**)&device_child_node_array, totalChildNodesSize);
    checkCudaError ( "Node array hipMalloc booommm!!!!" );

    hipMemcpy (device_child_node_array, host_child_node_array, totalChildNodesSize, hipMemcpyHostToDevice);
    checkCudaError ( "Node array memCopy booommm!!!!" );

    // Convert road polygon to array
    std::vector<mp::Vec2D> road_polygon = getRoadPolygon (overall_info->road_info);
    mp::Vec2D* host_road_polygon = road_polygon.data();

    uint totalRoadPolygonPoints = road_polygon.size();
    uint totalRoadPolygonSize = totalRoadPolygonPoints * sizeof(mp::Vec2D);

    mp::Vec2D* device_road_polygon;

    hipMalloc ((void**)&device_road_polygon, totalRoadPolygonSize);
    checkCudaError ( "Road polygon hipMalloc booommm!!!!" );

    hipMemcpy (device_road_polygon, host_road_polygon, totalRoadPolygonSize, hipMemcpyHostToDevice);
    checkCudaError ( "Road polygon memCopy booommm!!!!" );

    // Convert traffic polygons to array
    std::vector<mp::Vec2D> traffic_polygons = getTrafficPolygons (overall_info->traffic);

    mp::Vec2D* host_traffic_polygons_array = traffic_polygons.data();

    uint total_traffic_vehicles = overall_info->traffic.size();
    uint total_traffic_polygons_size = total_traffic_vehicles * 4 * sizeof (mp::Vec2D);

    mp::Vec2D* device_traffic_polygons_array;

    hipMalloc ((void**)&device_traffic_polygons_array, total_traffic_polygons_size);
    checkCudaError ( "Traffic polygons array hipMalloc booommm!!!!" );

    hipMemcpy (device_traffic_polygons_array, host_traffic_polygons_array, total_traffic_polygons_size, hipMemcpyHostToDevice);
    checkCudaError ( "Traffic polygons array memCopy booommm!!!!" );

    double lane_center_y = overall_info->nearest_lane_point_with_index.second.y;

    std::size_t num_blocks = static_cast<std::size_t>( ceil( child_nodes.size()/config.threads_per_block ) );

    // Kernel call
    calculate_cost <<<num_blocks, config.threads_per_block>>> (config.threads_per_block,
                                               device_child_node_array,
                                               device_road_polygon, totalRoadPolygonPoints,
                                               device_traffic_polygons_array,
                                               total_traffic_vehicles,
                                               lane_center_y,
                                               config.dist_to_goal,
                                               overall_info->ego_state->pose.x,
                                               overall_info->ego_state->pose.y);
    hipDeviceSynchronize();

    // Copy all necessary data from GPU to CPU
    hipMemcpy (host_child_node_array, device_child_node_array, totalChildNodesSize, hipMemcpyDeviceToHost);

    // Free memory on the gpu
    hipFree (device_child_node_array);
    hipFree (device_road_polygon);
    hipFree (device_traffic_polygons_array);

    // Store nodes evaluated in the recent cycle
    overall_info->mp_info.curr_eval_nodes.clear();
    for (uint i = 0; i < totalChildNodes; ++i)
    {
        overall_info->mp_info.curr_eval_nodes.push_back (host_child_node_array[i]);
    }
}

std::vector<mp::Vec2D> getRoadPolygon(const mp::RoadInfo& road_info)
{
    std::vector<mp::Vec2D> polygon_points;

    mp::LaneInfo leftmost_lane = road_info.lanes[0];
    double leftmost_lane_half_width = leftmost_lane.lane_width / 2;

    for ( std::vector<mp::Pose2D>::const_iterator iter = leftmost_lane.lane_points.begin();
          iter != leftmost_lane.lane_points.end(); ++iter )
    {
        mp::Vec2D point;
        point.x = leftmost_lane_half_width * cos(iter->theta + M_PI_2) + iter->x;
        point.y = leftmost_lane_half_width * sin(iter->theta + M_PI_2) + iter->y;

        polygon_points.push_back( point );
    }

    // Get rightmost lane edge
    mp::LaneInfo rightmost_lane = road_info.lanes[ road_info.num_lanes - 1 ];
    double rightmost_lane_half_width = rightmost_lane.lane_width / 2;

    for ( std::vector<mp::Pose2D>::const_reverse_iterator iter = rightmost_lane.lane_points.rbegin();
          iter != rightmost_lane.lane_points.rend(); ++iter )
    {
        mp::Vec2D point;
        point.x = rightmost_lane_half_width * cos(iter->theta - M_PI_2) + iter->x;
        point.y = rightmost_lane_half_width * sin(iter->theta - M_PI_2) + iter->y;

        polygon_points.push_back( point );
    }

    return polygon_points;
}

std::vector<mp::Vec2D> getTrafficPolygons(const std::vector<mp::Vehicle>& traffic)
{
    std::vector<mp::Vec2D> traffic_polygons;

    for (const auto& vehicle : traffic)
    {
        double yaw = vehicle.pose.theta;

        double front      = 2.389;    // vehicle.length/2 (since simulator publishes midpoint as pose)
        double rear       = 2.389;    // vehicle.length/2
        double width_by_2 = 1.05;

        mp::Vec2D front_left;
        front_left.x = cos(yaw)*front + cos(yaw + M_PI_2)*width_by_2 + vehicle.pose.x;
        front_left.y = sin(yaw)*front + sin(yaw + M_PI_2)*width_by_2 + vehicle.pose.y;

        mp::Vec2D front_right;
        front_right.x = cos(yaw)*front + cos(yaw - M_PI_2)*width_by_2 + vehicle.pose.x;
        front_right.y = sin(yaw)*front + sin(yaw - M_PI_2)*width_by_2 + vehicle.pose.y;

        mp::Vec2D rear_left;
        rear_left.x = - cos(yaw)*rear + cos(yaw + M_PI_2)*width_by_2 + vehicle.pose.x;
        rear_left.y = - sin(yaw)*rear + sin(yaw + M_PI_2)*width_by_2 + vehicle.pose.y;

        mp::Vec2D rear_right;
        rear_right.x = - cos(yaw)*rear + cos(yaw - M_PI_2)*width_by_2 + vehicle.pose.x;
        rear_right.y = - sin(yaw)*rear + sin(yaw - M_PI_2)*width_by_2 + vehicle.pose.y;

        traffic_polygons.push_back (rear_left);
        traffic_polygons.push_back (front_left);
        traffic_polygons.push_back (front_right);
        traffic_polygons.push_back (rear_right);
    }

    return traffic_polygons;
}


}
