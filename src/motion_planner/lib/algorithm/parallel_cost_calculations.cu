#include "hip/hip_runtime.h"
#include "parallel_cost_calculations.cuh"

namespace cuda_mp
{

__global__
void calculate_cost()
{
    printf("\n Thread id printing from GPU: %d ", threadIdx.x);
}

void calculateCost(mp::Node* node, const mp::PlannerConfig& config)
{
    std::vector<mp::Node> child_nodes = mp::getChildNodes(node, config);

    calculate_cost <<<2,2>>> ();
}

}
